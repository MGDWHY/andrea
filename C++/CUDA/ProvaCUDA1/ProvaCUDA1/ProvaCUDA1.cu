#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include "hip/hip_runtime.h"
#include <ctime>

using namespace std;

#define NUM_ELEMENTS 512 * 1000

__global__ void vecAddDevice(float * A, float * B, float * C) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	C[i] = A[i] + B[i];
}

int main() {
	
	float * hA, * hB, * hC;
	float * dA, * dB, * dC;
	int size = NUM_ELEMENTS * sizeof(float);
	int device;
	char ch;
	hipDeviceProp_t deviceProperties;
	
	hA = new float[NUM_ELEMENTS];
	hB = new float[NUM_ELEMENTS];
	hC = new float[NUM_ELEMENTS];

	// get device properties
	hipGetDevice(&device);
	hipGetDeviceProperties(&deviceProperties, device);
	cout << "Multiprocessors count: " << deviceProperties.multiProcessorCount << endl;
	cout << "Warp size: " << deviceProperties.warpSize << endl;
	cout << "Max Threads per Block: " << deviceProperties.maxThreadsPerBlock << endl;
	
	int numBlocks = NUM_ELEMENTS / deviceProperties.maxThreadsPerBlock;
	int threadsPerBlock = deviceProperties.maxThreadsPerBlock;

	// init vectors
	for(int i = 0; i < NUM_ELEMENTS; i++) {
		hA[i] = rand() / (float) RAND_MAX;
		hB[i] = rand() / (float) RAND_MAX;
		hC[i] = 0.0f;
	}

	cout << "Allocate device memory..." << endl;
	
	// allocate device memory
	hipMalloc(&dA, size);
	hipMalloc(&dB, size);
	hipMalloc(&dC, size);
	
	// copy data to device memory
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
	
	cout << "Starting kernel..." << endl <<
		"Blocks: " << numBlocks << endl <<
		"Threads per block: " << threadsPerBlock << endl;


	clock_t t1 = clock();

	vecAddDevice<<<numBlocks, threadsPerBlock>>>(dA, dB, dC);
	
	hipError_t e = hipDeviceSynchronize();

	if(e == hipSuccess)
		cout << "Done." << endl;
	else
		cout << "Error: " << hipGetErrorString(e) << endl;

	clock_t t2 = clock() - t1;
	
	double t = ((double)t2 / CLOCKS_PER_SEC * 1000.0);

	cout << "Time elapsed: " << t << " ms" << endl;

	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

	cout << "Freeing device memory..." << endl;
		
	// free device memory
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	

	cin >> ch;
	
	return 0;
}

