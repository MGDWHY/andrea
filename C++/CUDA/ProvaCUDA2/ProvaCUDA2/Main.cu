#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <ctime>
#include "gl/glew.h"
#include "gl/glut.h"
#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"

using namespace std;

#define W_LEFT -150.0f
#define W_RIGHT 150.0f
#define W_BOTTOM -150.0f
#define W_TOP 150.0f


#define GL_ERROR(x, i) std::cout << x << ": Error code -> " << i << std::endl
#define MSG(x) std::cout << (x) << std::endl

#define RND(i) ((float) rand() / RAND_MAX) * (i)

#define NUM_PARTICLES 512 * 15000
#define THREADS_PER_BLOCK 128
#define BLOCKS NUM_PARTICLES / THREADS_PER_BLOCK
#define TIME_SECS (float) clock() / CLOCKS_PER_SEC

#define ACTION_NONE 0
#define ACTION_ATTRACT 1
#define ACTION_REPULSE 2

#define G_CONSTANT 9.8f

#define SHOW_FPS

void InitGL();
void Render();
void Reshape(int, int);
void Keyboard(unsigned char, int, int);
void Mouse(int, int, int, int);
void MouseMotion(int, int);

void InitBuffers();
void InitShaders();
void InitCUDA();

void LastGLError(const char*);

void DrawParticles(float3);

char * LoadTextFile(const char *);

GLuint CreateShader(char*,int,GLenum);

struct Particle {
	float2 position;
	float2 velocity;
};

const float3 colors[5] = {
						make_float3(1.0, 0.2, 0.2),
						make_float3(0.7, 0.7, 0.2),
						make_float3(0.0, 0.3, 0.8),
						make_float3(0.8, 0.2, 0.8),
						make_float3(0.2, 1.0, 0.2)

					};

GLuint vao, vbo;
GLuint shaderProgram;

hipGraphicsResource * vboCuda;

int action = ACTION_NONE;
GLfloat actionX, actionY;

__global__ void EvolveParticles(Particle * particles, GLfloat dt, int action, GLfloat actionX, GLfloat actionY) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	
	// Load particle from global memory
	Particle p = particles[index];
	
	p.position.x += p.velocity.x * dt;
	p.position.y += p.velocity.y * dt;

	if(action == ACTION_ATTRACT) {
		float dx = actionX - p.position.x;
		float dy = actionY - p.position.y;
		float module = sqrtf(dx*dx + dy*dy);

		if(module < 1.0f) module = 1.0f;

		p.velocity.x += dx / module * G_CONSTANT * 8.0f / module;
		p.velocity.y += dy / module * G_CONSTANT * 8.0f / module;
	} else if(action == ACTION_REPULSE) {
		float dx = actionX - p.position.x;
		float dy = actionY - p.position.y;
		float module = sqrtf(dx*dx + dy*dy);

		if(module < 1.0f) module = 1.0f;

		p.velocity.x -= dx / module * G_CONSTANT * 60.0f / module;
		p.velocity.y -= dy / module * G_CONSTANT * 60.0f / module;
	}

	p.velocity.y -= dt * G_CONSTANT;

	if(p.position.y < W_BOTTOM && p.velocity.y < 0.0f) {
		p.velocity.y = - p.velocity.y * 0.5f;
		p.velocity.x = p.velocity.x * 0.8f;
	}
	else if(p.position.y > W_TOP && p.velocity.y > 0.0f)
		p.velocity.y = - p.velocity.y;

	else if(p.position.x < W_LEFT && p.velocity.x < 0.0f)
		p.velocity.x = - p.velocity.x * 0.8f;
	else if(p.position.x > W_RIGHT && p.velocity.x > 0.0f)
		p.velocity.x = - p.velocity.x * 0.8f;
	
	// Store updated particle on global memory
	particles[index] = p;
}


int main(int argc, char **argv) {
	
	// prima di TUTTO (TUTTO TUTTO TUTTO) creare la finestra senn� non funziona una mazza
	glutInit(&argc, argv);
	// rgba mode, double buffering, depth buffering, stencil buffering
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	// window' top left corner position
	glutInitWindowPosition(0,0);
	// window's size
	glutInitWindowSize(640, 480);
	// create window
	glutCreateWindow("CUDA Particles");
	// Finestra creata... Adesso dovrebbe andare, ma


	glewInit();
	
	InitGL();

	glutDisplayFunc(Render);
	glutReshapeFunc(Reshape);
	glutIdleFunc(Render);
	glutKeyboardFunc(Keyboard);
	glutMouseFunc(Mouse);
	glutMotionFunc(MouseMotion);

	//glutFullScreen();

	glutMainLoop();

}



// called when window is resized
void Reshape(int w, int h) {
	glViewport(0,0,w,h); // viewport resize
}


// called when window is drawn
void Render() {
	static float prevTime = TIME_SECS;
	static float dtsecs = 0.0f;
	static float fpsTimer = 0.0f;
	static float colorTimer = 0.0f;
	static float3 color1, color2, colorResult;

	static int colorIndex = 0;
	static int fps;

	dtsecs = TIME_SECS - prevTime;
	prevTime = TIME_SECS;

	colorTimer += dtsecs / 10.0f;
	
	if(colorTimer >= 2.0f) {
		colorTimer = 0.0f;
		if(colorIndex == 4)
			colorIndex = 0;
		else
			colorIndex++;
	}

	color1 = colors[colorIndex];
	color2 = colorIndex == 4 ? colors[0] : colors[colorIndex+1];
	
	if(colorTimer < 1.0f)
		colorResult = color1;
	else {
		float t = colorTimer - 1.0f;
		colorResult.x = color2.x * t + color1.x * (1-t);
		colorResult.y = color2.y * t + color1.y * (1-t);
		colorResult.z = color2.z * t + color1.z * (1-t);
	}

	#ifdef SHOW_FPS
		fpsTimer += dtsecs;
		fps++;
		
		if(fpsTimer > 1.0f) {
			std::cout << "FPS: " << fps << std::endl;
			fpsTimer -= 1.0f;
			fps = 0;
		}
	#endif

	Particle * particles;
	size_t length;

	hipGraphicsMapResources(1, &vboCuda, 0);
	hipGraphicsResourceGetMappedPointer((void**)&particles, &length, vboCuda);

	EvolveParticles<<<BLOCKS, THREADS_PER_BLOCK>>>(particles, dtsecs, action, actionX, actionY);

	hipDeviceSynchronize();

	hipGraphicsUnmapResources(1, &vboCuda, 0);

	DrawParticles(colorResult);

	glutSwapBuffers(); // swap backbuffer with frontbuffer
}


void InitGL() {
	// Init opengl(depth test, blending, lighting and so on...)
	glDisable(GL_DEPTH_TEST);
	glEnable(GL_BLEND);

	glBlendFunc(GL_SRC_ALPHA, GL_ONE); // additive blending

	InitBuffers();
	InitCUDA();
	InitShaders();

	LastGLError("InitGL():");
}


// Called by keyboard events
void Keyboard(unsigned char key, int x, int y) {
	if(key == 'q') {
		exit(0);
	}
}



// Called by mouse events
void Mouse(int button, int state, int x, int y) {
	if(state == GLUT_DOWN) {
		actionX = (GLfloat) x / glutGet(GLUT_WINDOW_WIDTH) * (W_RIGHT - W_LEFT) + W_LEFT;
		actionY = (W_TOP - W_BOTTOM) - (GLfloat) y / glutGet(GLUT_WINDOW_HEIGHT) * (W_TOP - W_BOTTOM) + W_BOTTOM;		
		if(button == GLUT_LEFT)
			action = ACTION_ATTRACT;
		else
			action = ACTION_REPULSE;
	} else
		action = ACTION_NONE;
}

void MouseMotion(int x, int y) {
	if(action != ACTION_NONE) {
		actionX = (GLfloat) x / glutGet(GLUT_WINDOW_WIDTH) * (W_RIGHT - W_LEFT) + W_LEFT;
		actionY = (W_TOP - W_BOTTOM) - (GLfloat) y / glutGet(GLUT_WINDOW_HEIGHT) * (W_TOP - W_BOTTOM) + W_BOTTOM;		
	}
}

void InitBuffers() {

	Particle * particles = new Particle[NUM_PARTICLES];

	for(int i = 0; i < NUM_PARTICLES; i++) {
		particles[i].position.x = RND(W_RIGHT - W_LEFT) + W_LEFT;
		particles[i].position.y = RND(W_TOP - W_BOTTOM) + W_BOTTOM;

		particles[i].velocity.x = RND(20.0f) - 10.0f;
		particles[i].velocity.y = RND(20.0f) - 10.0f;
	}

	glGenVertexArrays(1, &vao);
	glGenBuffers(1, &vbo);

	glBindVertexArray(vao);
	glBindBuffer(GL_ARRAY_BUFFER, vbo);

	glBufferData(GL_ARRAY_BUFFER, sizeof(Particle) * NUM_PARTICLES, (void*) particles, GL_STATIC_DRAW);

	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 16, 0);
	glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 16, (char*)NULL + 8);

	glEnableVertexAttribArray(0);
	//glEnableVertexAttribArray(1);

	glBindVertexArray(0);
	
	LastGLError("InitBuffers()");
}

void InitCUDA() {
	cudaGLSetGLDevice(0);
	hipGraphicsGLRegisterBuffer(&vboCuda, vbo, cudaGraphicsMapFlagsNone);
}


void DrawParticles(float3 color) {

	static float tmatrix[16];

	glClear(GL_COLOR_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);	
	glLoadIdentity();
	glOrtho(W_LEFT, W_RIGHT, W_BOTTOM, W_TOP, -1, 1);

	glUseProgram(shaderProgram);

	int locMat = glGetUniformLocation(shaderProgram, "in_ModelViewProjectionMatrix");
	int locColor = glGetUniformLocation(shaderProgram, "in_Color");

	glGetFloatv(GL_PROJECTION_MATRIX, tmatrix);

	glUniformMatrix4fv(locMat, 1, GL_FALSE, tmatrix);
	glUniform4f(locColor, color.x, color.y, color.z, 0.1f);

	glBindVertexArray(vao);

	glDrawArrays(GL_POINTS, 0, NUM_PARTICLES);

	glBindVertexArray(0);

	glUseProgram(0);
}


void LastGLError(const char * msg) {
	GLuint error = glGetError();
	if(error != GL_NO_ERROR)
		GL_ERROR(msg, error);
}

char * LoadTextFile(const char * fileName, int * length) {
	ifstream file;
	char * data = NULL;
	int len;

	file.open(fileName, ifstream::binary);
	
	file.seekg(0, ios_base::end);
	len = file.tellg();
	file.seekg(0, ios_base::beg);

	data = new char[len];

	if(!file.eof())
		file.read(data, len);
	else
		MSG("Bad file!");

	

	file.close();

	*length = len;

	return data;
}

void InitShaders() {
	GLuint vs, gs, fs;
	int vsl, gsl, fsl;
	char *vsSrc, *gsSrc, *fsSrc;
	
	vsSrc = LoadTextFile("particles.vert", &vsl);
	gsSrc = LoadTextFile("particles.geom", &gsl);
	fsSrc = LoadTextFile("particles.frag", &fsl);

	vs = CreateShader(vsSrc, vsl, GL_VERTEX_SHADER);
	gs = CreateShader(gsSrc, gsl, GL_GEOMETRY_SHADER);
	fs = CreateShader(fsSrc, fsl, GL_FRAGMENT_SHADER);

	shaderProgram = glCreateProgram();

	glAttachShader(shaderProgram, vs);
	glAttachShader(shaderProgram, gs);
	glAttachShader(shaderProgram, fs);
	
	glLinkProgram(shaderProgram);

	int linkStatus = 0;

	glGetProgramiv(shaderProgram, GL_LINK_STATUS, &linkStatus);
	if(linkStatus)
		MSG("Program linked!");
	else
		MSG("Can't link the program!");
}

GLuint CreateShader(char * source, int length, GLenum type) {
	GLuint shader = glCreateShader(type);

	glShaderSource(shader, 1, (const char**) &source, &length);

	glCompileShader(shader);

	int compiled = 0;
	glGetShaderiv(shader, GL_COMPILE_STATUS, &compiled);
	if(compiled)
		MSG("Shader compiled!");
	else	
		MSG("Shader not compiled!");

	return shader;
}